/**************/
/* TIMING GPU */
/**************/

#include "TimingGPU.cuh"
#include "cudaErrorCheck.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

struct PrivateTimingGPU {
	hipEvent_t     start;
	hipEvent_t     stop;
};

// default constructor
TimingGPU::TimingGPU() { privateTimingGPU = new PrivateTimingGPU; }

// default destructor
TimingGPU::~TimingGPU() { }

void TimingGPU::StartCounter()
{
	CudaSafeCall(hipEventCreate(&((*privateTimingGPU).start)));
	CudaSafeCall(hipEventCreate(&((*privateTimingGPU).stop)));
	CudaSafeCall(hipEventRecord((*privateTimingGPU).start, 0));
}

void TimingGPU::StartCounterFlags()
{
	int eventflags = hipEventBlockingSync;

	CudaSafeCall(hipEventCreateWithFlags(&((*privateTimingGPU).start), eventflags));
	CudaSafeCall(hipEventCreateWithFlags(&((*privateTimingGPU).stop), eventflags));
	CudaSafeCall(hipEventRecord((*privateTimingGPU).start, 0));
}

// Gets the counter in ms
float TimingGPU::GetCounter()
{
	float   time;
	CudaSafeCall(hipEventRecord((*privateTimingGPU).stop, 0));
	CudaSafeCall(hipEventSynchronize((*privateTimingGPU).stop));
	CudaSafeCall(hipEventElapsedTime(&time, (*privateTimingGPU).start, (*privateTimingGPU).stop));
	return time;
}